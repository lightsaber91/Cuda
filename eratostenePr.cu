
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define THREADS 256
#define MINARG 2
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_ERROR( hipEventCreate(start));
	HANDLE_ERROR( hipEventCreate(stop));
	HANDLE_ERROR( hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_ERROR( hipEventRecord(*stop, 0));
	HANDLE_ERROR( hipEventSynchronize(*stop));
	float time=0;
	HANDLE_ERROR( hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f\n", time);
	HANDLE_ERROR( hipEventDestroy(*start));
	HANDLE_ERROR( hipEventDestroy(*stop));
}

void print(int *array, int size){
    int i =0;
    int c = 0;
    for (i=0;i<size;i++){
        if (array[i]) {
            printf("%d\n", i+1);
            c++;
        }
    }
    printf("Total number of primes: %d\n", c);
}

__global__ void eliminateMultiples(int *list, int end, int next, int fine) {
}

void findAllPrimeNumbers(int N){
	if(N%2) {
        N+=1;
    }
    int blocks = (((N-2)/2)+(THREADS-1))/THREADS;
    int *dev_list, *dev_next;
    int *list = new int[N];
    int next;
	int i;
    hipEvent_t start,stop;
	printf("Number of threads: %d, Number of blocks: %d\n",THREADS,blocks);
    //cudaMalloc((void**)&dev_end, sizeof(int));
    hipMalloc((void**)&dev_next,sizeof(int));
    hipMalloc((void**)&dev_list,sizeof(int)*N);
	for(i=0; i<N; i++) {
		list[i]=1;
	}
    hipMemcpy(dev_list,list, sizeof(int)*N, hipMemcpyHostToDevice);
    next=2;
    hipMemcpy(dev_next, &next, sizeof(int), hipMemcpyHostToDevice);
    int fine = (int) (sqrt(N)+0.5);
    startTimer(&start,&stop);
    eliminateMultiples<<<blocks,THREADS>>>(dev_list, N, next, fine);
    hipDeviceSynchronize();
    stopAndPrint(&start,&stop);
    hipMemcpy(list, dev_list, sizeof(int)*N, hipMemcpyDeviceToHost);
    hipFree(dev_list);
    print(list, N);
    delete[] list;
}

int main(int argc, char *argv[]) {
	if(argc<MINARG) {
		fprintf(stderr,"Usage: %s N\n",argv[0]);
		exit(-1);
	}
    int N = atoi(argv[1]);
	if(N<0) {
		fprintf(stderr,"Invalid number: %d must be > 0\n",N);
		exit(-1);
	}
    printf("Prime numbers from 0 to %d:\n",N);
    findAllPrimeNumbers(N);
    return 0;
}
