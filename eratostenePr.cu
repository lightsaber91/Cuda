
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define THREADS 256
#define MINARG 2
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_ERROR( hipEventCreate(start));
	HANDLE_ERROR( hipEventCreate(stop));
	HANDLE_ERROR( hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_ERROR( hipEventRecord(*stop, 0));
	HANDLE_ERROR( hipEventSynchronize(*stop));
	float time=0;
	HANDLE_ERROR( hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f milliseconds\n", time);
	HANDLE_ERROR( hipEventDestroy(*start));
	HANDLE_ERROR( hipEventDestroy(*stop));
}

void print(int *array, int size){
    int i = 0;
    int c = 0;
    for (i=0;i<size;i++){
        if (array[i]) {
            printf("%d\n", array[i]);
            c++;
        }
    }
    printf("Total number of primes: %d\n", c);
}

__global__ void eliminateMultiples(int *list, int end, int *next, int fine) {
    __shared__ unsigned int block_next;
    block_next = *next;
    unsigned long start, i;
    do {
        start = (unsigned long) block_next*(threadIdx.x + 2 + blockIdx.x * blockDim.x) - 1;
        for(i = start; i < end; i += (unsigned long) block_next*blockDim.x*gridDim.x) {
            //elimino i multipli
            list[i] = 0;
        }
        __syncthreads();
        if(threadIdx.x == 0) {
            unsigned int j;
            bool found = false;
            //cambio il next
            if(block_next == 2) {
                j = block_next;
            }
            else
                j = block_next + 1;
            for(; j < end && found == false; j+=2) {
                if(list[j] > block_next) {
                    block_next = list[j];
                    found = true;
                }
            }
        }
        __syncthreads();
    }while(block_next < fine);
}

void findAllPrimeNumbers(int N){
	//Definisco il numero di blocchi
    if(N%2) {
        N+=1;
    }
    int blocks = (((N-2)/2)+(THREADS-1))/THREADS;
    printf("Number of threads: %d, Number of blocks: %d\n",THREADS,blocks);
    //Variabili GPU
    int *dev_list, *dev_next;
    //Variabili CPU
    int *list = new int[N];
    int next = 2;
    for(int i=0; i<N; i++) {
        list[i]=i+1;
    }
    int fine = (int) (sqrt(N)+0.5);
    //Timer
    hipEvent_t start,stop;
    //Allocazione su GPU
    //cudaMalloc((void**)&dev_end, sizeof(int));
    hipMalloc((void**)&dev_next,sizeof(int));
    hipMalloc((void**)&dev_list,sizeof(int)*N);
    //Copia dati sulla GPU
    hipMemcpy(dev_list,list, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_next, &next, sizeof(int), hipMemcpyHostToDevice);
    //Inizializzazione del Timer
    startTimer(&start,&stop);
    //Lancio del Kernel
    eliminateMultiples<<<blocks,THREADS>>>(dev_list, N, dev_next, fine);
    hipDeviceSynchronize();
    //Fine del timer
    stopAndPrint(&start,&stop);
    //Ricopio il risultato sulla GPU
    hipMemcpy(list, dev_list, sizeof(int)*N, hipMemcpyDeviceToHost);
    //Libero Memoria
    hipFree(dev_list);
    hipFree(dev_next);
    //Stampo informazioni
    print(list, N);
    delete[] list;
}

int main(int argc, char *argv[]) {
	if(argc<MINARG) {
		fprintf(stderr,"Usage: %s N\n",argv[0]);
		exit(-1);
	}
    int N = atoi(argv[1]);
    if(N<0) {
		fprintf(stderr,"Invalid number: %d must be > 0\n",N);
		exit(-1);
	}
    printf("Prime numbers from 0 to %d:\n",N);
    findAllPrimeNumbers(N);
    return 0;
}
