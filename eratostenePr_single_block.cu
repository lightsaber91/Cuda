
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define THREADS 256
#define MINARG 2
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))
static void HandleError( hipError_t err, const char *file, int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}

void startTimer(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_ERROR( hipEventCreate(start));
	HANDLE_ERROR( hipEventCreate(stop));
	HANDLE_ERROR( hipEventRecord(*start, 0));
}

void stopAndPrint(hipEvent_t *start, hipEvent_t *stop) {
	HANDLE_ERROR( hipEventRecord(*stop, 0));
	HANDLE_ERROR( hipEventSynchronize(*stop));
	float time=0;
	HANDLE_ERROR( hipEventElapsedTime(&time, *start, *stop));
	printf("Elapsed Time: %f in milliseconds\n", time);
	HANDLE_ERROR( hipEventDestroy(*start));
	HANDLE_ERROR( hipEventDestroy(*stop));
}

void print(int *array, int size){
    int i =0;
    int c = 0;
    for (i=0;i<size;i++){
        if (array[i]) {
//            printf("%d\n", i+1);
            c++;
        }
    }
    printf("Total number of primes: %d\n", c);
}

__global__ void eliminateMultiples(int *list, int end, int *next, int fine) {
    //caso di un singolo blocco
    unsigned int start;
    do {
        start = (*next)*(threadIdx.x + 2);
        for(int i = start-1; i < end; i += (*next)*blockDim.x) {
            //elimino i multipli
            list[i] = 0;
        }
        __syncthreads();
        if(threadIdx.x == 0) {
            unsigned int j;
	    bool found = false;
            //cambio il next
	    if(*next == 2) {
		j = *next;
	    }
	    else
		j = *next + 1;
            for(; j < end && found == false; j+=2) {
                if(list[j] > *next) {
                    *next = list[j];
                    found = true;
                }
            }
        }
        __syncthreads();
    } while(*next < fine);
}

void findAllPrimeNumbers(int N){
	//Definisco il numero di blocchi
    if(N%2) {
        N+=1;
    }
    //int blocks = (((N-2)/2)+(THREADS-1))/THREADS;
    int blocks = 1;
    //Variabili GPU
    int *dev_list, *dev_next;
    //Variabili CPU
    int *list = new int[N];
    int next = 2;
    for(int i=0; i<N; i++) {
        list[i]=i+1;
    }
    int fine = (int) (sqrt(N)+0.5);
    //Timer
    hipEvent_t start,stop;
    //Allocazione su GPU
    //cudaMalloc((void**)&dev_end, sizeof(int));
    hipMalloc((void**)&dev_next,sizeof(int));
    hipMalloc((void**)&dev_list,sizeof(int)*N);
    //Copia dati sulla GPU
    hipMemcpy(dev_list,list, sizeof(int)*N, hipMemcpyHostToDevice);
    hipMemcpy(dev_next, &next, sizeof(int), hipMemcpyHostToDevice);
    //Inizializzazione del Timer
    startTimer(&start,&stop);
    //Lancio del Kernel
    eliminateMultiples<<<blocks,THREADS>>>(dev_list, N, dev_next, fine);
    hipDeviceSynchronize();
    //Fine del timer
    stopAndPrint(&start,&stop);
    //Ricopio il risultato sulla GPU
    hipMemcpy(list, dev_list, sizeof(int)*N, hipMemcpyDeviceToHost);
//DEBUG
    hipMemcpy(&next, dev_next, sizeof(int), hipMemcpyDeviceToHost);
    printf("next := %d\n", next);
//FINE DEBUG
    //Libero Memoria
    hipFree(dev_list);
    hipFree(dev_next);
    //Stampo informazioni
    printf("Number of threads: %d, Number of blocks: %d\n",THREADS,blocks);
    print(list, N);
    delete[] list;
}

int main(int argc, char *argv[]) {
	if(argc<MINARG) {
		fprintf(stderr,"Usage: %s N\n",argv[0]);
		exit(-1);
	}
    int N = atoi(argv[1]);
	if(N<0) {
		fprintf(stderr,"Invalid number: %d must be > 0\n",N);
		exit(-1);
	}
    printf("Prime numbers from 0 to %d:\n",N);
    findAllPrimeNumbers(N);
    return 0;
}
